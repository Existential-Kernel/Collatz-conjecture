#ifndef KERNEL
#define KERNEL
#include <iostream>
#include <hip/hip_runtime.h>
#include <map>
#include <functional>
#include "defs.hpp"

__host__ void Collatz(ULL n) {
    while (n > 1) {
        n % 2 == 0 ? n /= 2 : n = n * 3 + 1;
    }
    std::cout << 1 << std::endl;
}

__host__ void VerboseCollatz(ULL n) {
    std::cout << n << std::endl;
    while (n > 1) {
        auto m = n % 2 == 0;
        n = m * (n / 2) + !m * (n * 3 + 1);
        std::cout << n << std::endl;
    }
}

__host__ bool VerifyCollatz(ULL n) {
    try {
        n /= (n&-n);
        while(n > 1) {
            n = 3*n+1;
            n /= (n&-n);
        }
        return true;
    } catch(...) {
        return false;
    }
}

__device__ void ParallelCollatz(ULL n) {
    
}

__host__ LL CollatzCount(ULL n) {
    ULL count = 1;
    while (n > 1) {
        auto m = n % 2 == 0;
        n = m * (n / 2) + !m * (n * 3 + 1);
        count++;
    }
    return count;
}

#endif