#ifndef KERNEL
#define KERNEL
#include <iostream>
#include <hip/hip_runtime.h>
#include <map>
#include <functional>
#include "defs.cuh"

__host__ void Collatz(ULL n) {
    while (n > 1) {
        n % 2 == 0 ? n /= 2 : n = n * 3 + 1;
    }
    std::cout << 1 << std::endl;
}

__host__ void VerboseCollatz(ULL n) {
    while (n > 1) {
        n % 2 == 0 ? n /= 2 : n = n * 3 + 1;
        std::cout << n << std::endl;
    }
}

__host__ void VerifyCollatz(ULL n) {

}
__host__ void Catch(){
    try {

    } catch (const std::overflow_error& err) {
        std::cerr << "The program has experienced an overflow error!\n" << std::endl;
    } catch (const std::runtime_error& err) {
        std::cerr << "The program has experienced an underflow error!\n" << std::endl;
    } catch (const std::exception& err) {
        std::cerr << "The program has experienced an exeption error!\n" << std::endl;
    } catch (...) {
        std::cerr << "The program has experienced an unexpected error!\n" << std::endl;
    }
}

__device__ void ParallelCollatz(ULL n) {
    
}

__host__ ULL CollatzCount(ULL n) {
    ULL count = 1;
    while (n > 1) {
        n % 2 == 0 ? n /= 2 : n = n * 3 + 1;
        count++;
    }
    return count;
}

std::map<std::string, std::function<void(unsigned long long)>> funcMap = {
    { "Collatz", Collatz},
    { "VerboseCollatz", VerboseCollatz},
    { "VerifyCollatz", VerifyCollatz},
    { "CollatzCount", CollatzCount},
    { "ParallelCollatz", ParallelCollatz}
};

#endif