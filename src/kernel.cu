#ifndef KERNEL
#define KERNEL
#include <iostream>
#include <hip/hip_runtime.h>
#include <map>
#include <functional>
#include <bit>
#include <bitset>
#include <cstdint>
#include "defs.hpp"

__host__ void Collatz(ULL n) {
    std::cout << n << std::endl;
    while (n > 1) {
        n % 2 == 0 ? n /= 2 : n = n * 3 + 1;
    }
}

__host__ void VerboseCollatz(ULL n) {
    std::cout << n << std::endl;
    while (n > 1) {
        auto m = n % 2 == 0;
        n = m * (n / 2) + !m * (n * 3 + 1);
        std::cout << n << "\n";
    }
}

__host__ bool VerifyCollatz(ULL n) {
    try {
        n /= (n&-n);
        while(n > 1) {
            n = 3*n+1;
            n /= (n&-n);
        }
        /* For anybody reading this, the above expression is the equivalent to:
         *   if (n % 2 == 0) {  
         *     n = n / 2;
         *   } else {
         *     n = n * 3 + 1;
         *   }
        */
        return true;
    } catch(...) {
        return false;
    }
}

__device__ void ParallelCollatz(ULL n) {
    
}

__host__ ULL CollatzCount(ULL n) {
    ULL count = 1;
    while (n > 1) {
        auto m = n % 2 == 0;
        n = m * (n / 2) + !m * (n * 3 + 1);
        count++;
    }
    return count;
}

#endif