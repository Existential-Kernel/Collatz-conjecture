#ifndef KERNEL
#define KERNEL
#include <iostream>
#include <hip/hip_runtime.h>
#include <map>
#include <functional>
#include <bit>
#include <bitset>
#include <cstdint>
#include "defs.hpp"

__host__ inline void Collatz(ULL n) {
    std::cout << n << std::endl;
    while (n > 1) {
        n % 2 == 0 ? n /= 2 : n = n * 3 + 1;
    }
}

__host__ inline void VerboseCollatz(unsigned long long n) {
    std::cout << n << std::endl;
    while (n > 1) {
        auto m = (n & 1) != 0;
        n = m * (n >> 1) + !m * (n * 3 + 1);
        std::cout << n << "\n";
    }
}

__host__ inline bool VerifyCollatz(ULL n) {
    try {
        n /= (n&-n);
        std::cout << "test: " << n << std::endl;
        while(n > 1) {
            n = 3*n+1;
            n /= (n&-n);
            std::cout << "test: " << n << std::endl;
        }
        /* For anybody reading this, the above expression is the equivalent to:
         *   if (n % 2 == 0) {  
         *     n = n / 2;
         *   } else {
         *     n = n * 3 + 1;
         *   }
        */
        return true;
    } catch(...) {
        return false;
    }
}

__device__ inline void ParallelCollatz(ULL n) {
    
}

__host__ inline ULL CollatzCount(ULL n) {
    ULL count = 1;
    while (n > 1) {
        auto m = n % 2 == 0;
        n = m * (n >> 1) + !m * (n * 3 + 1);
        count++;
    }
    return count;
}

#endif